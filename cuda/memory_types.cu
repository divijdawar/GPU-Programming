#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// comparing transfer times in pinned vs pageable memory
__global__ void vectorAdd(float *a, float *b, float *c, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx < N) {
        c[idx] = a[idx] + b[idx]; 
    }
}

int main() {
    int N = 1 << 20; 
    size_t bytes = N * sizeof(float); 
    float elapsed_time;
    hipEvent_t start, stop;
    
    // Initialize CUDA timing events
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    printf("Vector size: %d elements (%lu MB)\n\n", N, bytes / (1024 * 1024));
    
    // pagable memory 
    float *a, *b, *c; 
    a = (float*)malloc(bytes);
    b = (float*)malloc(bytes);
    c = (float*)malloc(bytes);

    for (int i = 0; i < N; i++) {
        a[i] = (float)(rand() %100) / 10.0f; 
        b[i] = (float)(rand() %100) / 10.0f; 
    }

    float *d_a, *d_b, *d_c; 
    hipMalloc(&d_a, bytes); 
    hipMalloc(&d_b, bytes); 
    hipMalloc(&d_c, bytes); 

    hipEventRecord(start, 0);
    
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice); 
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice); 

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; 

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N); 

    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    printf("Pageable Memory - first 5 results:\n");
    for (int i = 0; i < 5; i++) {
        printf("%.2f", c[i]);
        if (i < 4) printf(", ");
    }
    printf("\n\n");
    // this takes about 4.91ms on an average on a 3070
    printf("Pageable Memory Transfer Time: %.2f ms\n\n", elapsed_time);
    
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c); 
    free(a); 
    free(b); 
    free(c);

    // pinned memory 
    float *pinned_a, *pinned_b, *pinned_c; 
    hipHostMalloc(&pinned_a, bytes, hipHostMallocDefault); 
    hipHostMalloc(&pinned_b, bytes, hipHostMallocDefault); 
    hipHostMalloc(&pinned_c, bytes, hipHostMallocDefault); 
    
    for (int i = 0; i < N; i++) {
        pinned_a[i] = (float)(rand() %100) / 10.0f; 
        pinned_b[i] = (float)(rand() %100) / 10.0f; 
    }

    float *d_pinned_a, *d_pinned_b, *d_pinned_c;
    
    hipMalloc((void**)&d_pinned_a, bytes); 
    hipMalloc((void**)&d_pinned_b, bytes); 
    hipMalloc((void**)&d_pinned_c, bytes); 

    hipEventRecord(start, 0);
    
    hipMemcpyAsync(d_pinned_a, pinned_a, bytes, hipMemcpyHostToDevice); 
    hipMemcpyAsync(d_pinned_b, pinned_b, bytes, hipMemcpyHostToDevice); 

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_pinned_a, d_pinned_b, d_pinned_c, N); 

    hipMemcpyAsync(pinned_c, d_pinned_c, bytes, hipMemcpyDeviceToHost); 
    hipDeviceSynchronize();
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    printf("Pinned Memory - first 5 results:\n");
    for (int i = 0; i < 5; i++) {
        printf("%.2f", pinned_c[i]);
        if (i < 4) printf(", ");
    }
    printf("\n\n");
    // this takes about 1.07ms on an average on a 3070, which is 4.4x faster than pageable memory
    printf("Pinned Memory Transfer Time: %.2f ms\n", elapsed_time);

    hipHostFree(pinned_a); 
    hipHostFree(pinned_b); 
    hipHostFree(pinned_c); 
    hipFree(d_pinned_a); 
    hipFree(d_pinned_b); 
    hipFree(d_pinned_c); 

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
