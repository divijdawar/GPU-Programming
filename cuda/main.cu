#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void atomicAdd(const float *input, float *result , int N);

int main() {
    int N = 1<< 24; 
    size_t size = N * sizeof(float); 
    
    // allocate host memory 
    float *h_input = (float*)malloc(size);
    float h_result = 0.0f;

    // Initialize host array with random values
    for (int i = 0; i < N; i++) {
        h_input[i] = (float)(rand() % 100) / 100.0f;  // Random float between 0 and 1
    }

    // Allocate device memory 
    float *d_input, *d_result; 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_result, sizeof(float));
    hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

    // Kernel launch (example configuration, adjust as needed)
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    atomicAdd<<<gridSize, blockSize>>>(d_input, d_result, N);
    hipDeviceSynchronize();

    hipEventRecord(start);
    
    // Kernel execution
    atomicAdd<<<gridSize, blockSize>>>(d_input, d_result, N);
    hipDeviceSynchronize();
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate GFLOPs: total number of floating point operations divided by execution time (in seconds)
    float gflops = (float)N / (milliseconds * 1e6f); // 1e6 converts ms to seconds and adds GFLOP scaling
    printf("Time: %f ms, GFLOPs: %f\n", milliseconds, gflops);

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
    printf("Sum result: %f\n", h_result);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(h_input);
    hipFree(d_input);
    hipFree(d_result);
    return 0;
}